#include "op/impl/cuda/cuda_kernel.h"
#include "op/impl/cuda/cuda_perform.h"
#include "./cuda_perform_system_include.h"
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand.h>
#include <limits>

namespace minerva {
namespace cuda {

static void FindConfiguration(size_t size, int& num_blocks, int& num_threads) {
  num_threads = 0;
  if(size <= 32)
    num_threads = 32;
  else if(size <= 64)
    num_threads = 64;
  else if(size <= 128)
    num_threads = 128;
  else if(size <= 256)
    num_threads = 256;
  else if(size <= 512)
    num_threads = 512;
  else
    num_threads = 1024;
  num_blocks = static_cast<int>((size + num_threads - 1) / num_threads);
  if (num_blocks < 0 || 128 < num_blocks) {
    num_blocks = 128;
  }
}

void CudaPerformDotMult(float* a, float* b, float* c, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(a, b, c, size, MultOp());
  CheckCudaError("CudaPerformDotMult");
}

void CudaPerformDotDiv(float* a, float* b, float* c, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(a, b, c, size, DivOp());
  CheckCudaError("CudaPerformDotDiv");
}

void CudaPerformAdd(float* a, float* b, float* c, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(a, b, c, size, SumOp());
  CheckCudaError("CudaPerformAdd");
  //float one = 1.0;
  //CUBLAS_CALL(hipblasScopy(handle, size, a, 1, c, 1));
  //CUBLAS_CALL(hipblasSaxpy(handle, size, &one, b, 1, c, 1));
}

void CudaPerformCopy(float* a, float* b, size_t size, hipblasHandle_t handle) {
  CUBLAS_CALL(hipblasScopy(handle, size, a, 1, b, 1));
}

void CudaPerformSub(float* a, float* b, float* c, size_t size, hipblasHandle_t handle) {
  float minus_one = -1.0;
  CUBLAS_CALL(hipblasScopy(handle, size, a, 1, c, 1));
  CUBLAS_CALL(hipblasSaxpy(handle, size, &minus_one, b, 1, c, 1));
}

void CudaPerformMatMult(float* a, float* b, float* c, int m, int n, int k, hipblasHandle_t handle) {
  float one = 1.0;
  float zero = 0.0;
  CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &one, a, m, b, k, &zero, c, m));
}

void CudaPerformScale(float* in_data, float* res_data, size_t size, float val, hipblasHandle_t handle) {
  CUBLAS_CALL(hipblasScopy(handle, size, in_data, 1, res_data, 1));
  CUBLAS_CALL(hipblasSscal(handle, size, &val, res_data, 1));
}

void CudaPerformTranspose(float* a, float* c, int m, int n, hipblasHandle_t handle) {
  float zero = 0.0;
  float one = 1.0;
  CUBLAS_CALL(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &one, a, m, &zero, c, n, c, n));
}

void CudaPerformConstAdd(float* in, float* out, float val, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, val, size, SumOp());
  CheckCudaError("CudaPerformConstAdd");
}

void CudaPerformLeftConstSub(float* in, float* out, float val, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, val, size, ReverseSubOp());
  CheckCudaError("CudaPerformLeftConstSub");
}

void CudaPerformLeftConstDiv(float* in, float* out, float val, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, val, size, ReverseDivOp());
  CheckCudaError("CudaPerformLeftConstDiv");
}

void CudaPerformNormAddOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SumOp());
  CheckCudaError("CudaPerformNormAddOnCol");
}

void CudaPerformNormSubOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SubOp());
  CheckCudaError("CudaPerformNormSubOnCol");
}

void CudaPerformNormMultOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, MultOp());
  CheckCudaError("CudaPerformNormMultOnCol");
}

void CudaPerformNormDivOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, DivOp());
  CheckCudaError("CudaPerformNormDivOnCol");
}

void CudaPerformNormAddOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SumOp());
  CheckCudaError("CudaPerformNormAddOnRow");
}

void CudaPerformNormSubOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SubOp());
  CheckCudaError("CudaPerformNormSubOnRow");
}

void CudaPerformNormMultOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, MultOp());
  CheckCudaError("CudaPerformNormMultOnRow");
}

void CudaPerformNormDivOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, DivOp());
  CheckCudaError("CudaPerformNormDivOnRow");
}

void CudaPerformReductionSumOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n, SumOp());
  CheckCudaError("CudaPerformReductionSumOnCol");
}

void CudaPerformReductionMaxOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n, MaxOp());
  CheckCudaError("CudaPerformReductionMaxOnCol");
}

void CudaPerformReductionSumOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnRowKernel<<<block, thread, 0, stream>>>(in, out, m, n, SumOp());
  CheckCudaError("CudaPerformReductionSumOnRow");
}

void CudaPerformReductionMaxOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnRowKernel<<<block, thread, 0, stream>>>(in, out, m, n, MaxOp());
  CheckCudaError("CudaPerformReductionMaxOnRow");
}

void CudaPerformMaxIndexOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformMaxIndexOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n);
  CheckCudaError("CudaPerformMaxIndexOnCol");
}

void CudaPerformMaxIndexOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformMaxIndexOnRowKernel << <block, thread, 0, stream>>>(in, out, m, n);
  CheckCudaError("CudaPerformMaxIndexOnRow");
}

void CudaPerformReshape(float* in, float* out, size_t size, hipStream_t stream) {
  CUDA_CALL(hipMemcpyAsync(out, in, size, hipMemcpyDefault, stream));
}

void CudaPerformElewiseExp(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, ExpOp());
  CheckCudaError("CudaPerformEleWiseExp");
}

void CudaPerformElewiseLn(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, LnOp());
  CheckCudaError("CudaPerformEleWiseLn");
}

void CudaPerformElewiseNegative(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, NegativeOp());
  CheckCudaError("CudaPerformEleWiseNegative");
}

void CudaPerformConvBackwardBias(float* top_diff, float* bias_diff, int num_images, int top_num_channels, int top_height, int top_width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t bias_diff_desc;
  hipdnnTensorDescriptor_t top_diff_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_diff_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_diff_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bias_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, top_num_channels, 1, 1));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, top_height, top_width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnConvolutionBackwardBias(handle, &one, top_diff_desc, top_diff, &zero, bias_diff_desc, bias_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_diff_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bias_diff_desc));
}

void CudaPerformInstanceSoftmaxForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &one, desc, bottom, &zero, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformChannelSoftmaxForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &one, desc, bottom, &zero, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformInstanceSoftmaxBackward(float* diff, float* top, float* bottom, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnSoftmaxBackward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &one, desc, top, desc, diff, &zero, desc, bottom));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformChannelSoftmaxBackward(float* diff, float* top, float* bottom, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnSoftmaxBackward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &one, desc, top, desc, diff, &zero, desc, bottom));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformSigmoidForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_SIGMOID, &one, desc, bottom, &zero, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformReluForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_RELU, &one, desc, bottom, &zero, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformTanhForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_TANH, &one, desc, bottom, &zero, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformSigmoidBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_SIGMOID, &one, desc, top, desc, top_diff, desc, bottom, &zero, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformReluBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_RELU, &one, desc, top, desc, top_diff, desc, bottom, &zero, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformTanhBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_TANH, &one, desc, top, desc, top_diff, desc, bottom, &zero, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformMaxPoolingForward(float* bottom, float* top, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {
  // Calculate the dimension after pooling
  int pooled_height = (bottom_height + 2 * pad_height - window_height + stride_vertical - 1) / stride_vertical + 1;
  if (0 <= (pooled_height - 1) * stride_vertical - bottom_height - pad_height) {
    --pooled_height;
  }
  int pooled_width = (bottom_width + 2 * pad_width - window_width + stride_horizontal - 1) / stride_horizontal + 1;
  if (0 <= (pooled_width - 1) * stride_horizontal - bottom_width - pad_width) {
    --pooled_width;
  }
  hipdnnTensorDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  hipdnnTensorDescriptor_t top_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_MAX, window_height, window_width, pad_height, pad_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, pooled_height, pooled_width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnPoolingForward(handle, pool_desc, &one, bottom_desc, bottom, &zero, top_desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bottom_desc));
}

void CudaPerformAveragePoolingForward(float* bottom, float* top, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {
  // Calculate the dimension after pooling
  int pooled_height = (bottom_height + 2 * pad_height - window_height + stride_vertical - 1) / stride_vertical + 1;
  if (0 <= (pooled_height - 1) * stride_vertical - bottom_height - pad_height) {
    --pooled_height;
  }
  int pooled_width = (bottom_width + 2 * pad_width - window_width + stride_horizontal - 1) / stride_horizontal + 1;
  if (0 <= (pooled_width - 1) * stride_horizontal - bottom_width - pad_width) {
    --pooled_width;
  }
  hipdnnTensorDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  hipdnnTensorDescriptor_t top_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING, window_height, window_width, pad_height, pad_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, pooled_height, pooled_width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnPoolingForward(handle, pool_desc, &one, bottom_desc, bottom, &zero, top_desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bottom_desc));
}

void CudaPerformMaxPoolingBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {
  // Calculate the dimension after pooling
  int pooled_height = (bottom_height + 2 * pad_height - window_height + stride_vertical - 1) / stride_vertical + 1;
  if (0 <= (pooled_height - 1) * stride_vertical - bottom_height - pad_height) {
    --pooled_height;
  }
  int pooled_width = (bottom_width + 2 * pad_width - window_width + stride_horizontal - 1) / stride_horizontal + 1;
  if (0 <= (pooled_width - 1) * stride_horizontal - bottom_width - pad_width) {
    --pooled_width;
  }
  hipdnnTensorDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  hipdnnTensorDescriptor_t top_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_MAX, window_height, window_width, pad_height, pad_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, pooled_height, pooled_width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnPoolingBackward(handle, pool_desc, &one, top_desc, top, top_desc, top_diff, bottom_desc, bottom, &zero, bottom_desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bottom_desc));
}

void CudaPerformAveragePoolingBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {
  // Calculate the dimension after pooling
  int pooled_height = (bottom_height + 2 * pad_height - window_height + stride_vertical - 1) / stride_vertical + 1;
  if (0 <= (pooled_height - 1) * stride_vertical - bottom_height - pad_height) {
    --pooled_height;
  }
  int pooled_width = (bottom_width + 2 * pad_width - window_width + stride_horizontal - 1) / stride_horizontal + 1;
  if (0 <= (pooled_width - 1) * stride_horizontal - bottom_width - pad_width) {
    --pooled_width;
  }
  hipdnnTensorDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  hipdnnTensorDescriptor_t top_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING, window_height, window_width, pad_height, pad_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, pooled_height, pooled_width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnPoolingBackward(handle, pool_desc, &one, top_desc, top, top_desc, top_diff, bottom_desc, bottom, &zero, bottom_desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bottom_desc));
}

void CudaPerformRandn(float* dst, size_t size, unsigned int seed, float mean, float var) {
  hiprandGenerator_t gen;
  CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
  CURAND_CALL(hiprandGenerateNormal(gen, dst, size, mean, var));
  CURAND_CALL(hiprandDestroyGenerator(gen));
}

void CudaPerformRandBernoulli(float* dst, size_t size, unsigned int seed, float p, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformRandBernoulliKernel<<<block, thread, 0, stream>>>(dst, size, seed, p);
  CheckCudaError(__func__);
}

void CudaPerformFill(float* dst, size_t size, float val, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformFillKernel<<<block, thread, 0, stream>>>(dst, size, val);
  CheckCudaError("CudaPerformFill");
}

void CudaPerformSelect(float* dst, float* src, std::vector<int> indices, size_t cols, size_t rows, hipStream_t stream) {
  int block, thread;
  int size = cols * rows;
  int* indices_ptr;
  FindConfiguration(size, block, thread);
  CUDA_CALL(hipMalloc(&indices_ptr, indices.size() * sizeof(int)));
  CUDA_CALL(hipMemcpyAsync(indices_ptr, &indices[0], indices.size() * sizeof(int), hipMemcpyDefault, stream));
  SelectKernel<<<block, thread, 0, stream>>>(dst, src, &indices[0], cols, rows, indices.size());
  CheckCudaError("Select");
}

}  // namespace cuda
}  // namespace minerva

